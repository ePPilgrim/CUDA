//
// kernel routine
//
#include "hip/hip_runtime.h"

__global__ void my_first_kernel(float *x)
{
	// Uncomment line below and define integer "tid" as global index to vector "x"
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// Uncomment line below and define x[tid] to be equal to the thread index
	x[tid] = (float)threadIdx.x;
}

