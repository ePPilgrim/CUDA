#include "hip/hip_runtime.h"
#include <ctime>
#include<cmath>
#include<iostream>
#include<fstream>
#include<random>
#include<vector>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

float* genVects(int col, int row);
float* findMeans(float* vects, int col, int row);
float* findCovVals(float* vects, int col, int row);

const int BlckSzX = 32;
const int BlckSzY = 16;
const int BlckSz = 16;
const int LogN = 4;

__global__ void mean_kern(float* vects, int row, int col,float *sum_odata, int slotcnt)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	__shared__ float sum[BlckSzX][BlckSzX];
	sum[threadIdx.y][threadIdx.x] = vects[y * col + x];
	__syncthreads();

	int k = blockDim.x >> 1;
	for (int i = 0; i < LogN; ++i) {
		if (threadIdx.x < k) {
			sum[threadIdx.y][threadIdx.x] += sum[threadIdx.y][(k << 1) - threadIdx.x - 1];
			k >>= 1;
		}
		__syncthreads();
	}
	if (threadIdx.x == 0) sum_odata[blockIdx.y * slotcnt + blockIdx.x] = sum[threadIdx.y][0];
}

__global__ void var_kern(float* vects, int row, int col, float *sum_odata, int slotcnt)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (x == 0) {
		float sum = 0.0f;
		float* subsum = &sum_odata[y * slotcnt];
		for (int i = 0; i < slotcnt; ++i) {
			sum += subsum[i];
		}
		sum_odata[y*slotcnt] = sum / (float)col;
	}
	__syncthreads();
	//vects[y * col + x] -= sum_odata[y*slotcnt];	
}

__global__ void VarianceMatrix(const float* vects, int row, int col, float* out,const int slotsize) {
	int x = blockIdx.x * blockDim.x;
	x += threadIdx.x;
	int y = blockIdx.y * blockDim.y;
	y += threadIdx.y;
	int shift = blockIdx.x * slotsize;
	shift += y;

	__shared__ float outmat[BlckSzY][BlckSzX];
	__shared__ float subvects[BlckSzY][BlckSzX];
	subvects[threadIdx.y][threadIdx.x] = vects[y * col + x];
	__syncthreads();
	for(int i = 0; i < blockDim.y; ++ i)
	{
		if ((threadIdx.y + i) < blockDim.y) {
			outmat[threadIdx.y][threadIdx.x] = vects[y * col + x] * subvects[threadIdx.y + i][threadIdx.x];
		}
		__syncthreads();
		if ((threadIdx.y + i) < blockDim.y) {
			if (threadIdx.x < 16)  outmat[threadIdx.y][threadIdx.x] += outmat[threadIdx.y][threadIdx.x + 16];
			__syncthreads();
			if (threadIdx.x < 8)  outmat[threadIdx.y][threadIdx.x] += outmat[threadIdx.y][threadIdx.x + 8];
			__syncthreads();
			if (threadIdx.x < 4)  outmat[threadIdx.y][threadIdx.x] += outmat[threadIdx.y][threadIdx.x + 4];
			__syncthreads();
			if (threadIdx.x < 2)  outmat[threadIdx.y][threadIdx.x] += outmat[threadIdx.y][threadIdx.x + 2];
			__syncthreads();
			if (threadIdx.x == 0) {
				outmat[threadIdx.y][0] += outmat[threadIdx.y][1];
				out[shift] = outmat[threadIdx.y][0];
				shift += row - i;
			}
		}
		__syncthreads();
	}

	for (int i = blockDim.y; i < blockDim.y * (gridDim.y - blockIdx.y); i = i + blockDim.y) {
		subvects[threadIdx.y][threadIdx.x] = vects[(i + y) * col + x];
		__syncthreads();
		for (int j = 0; j < BlckSzY; ++j) {
			outmat[threadIdx.y][threadIdx.x] = vects[y * col + x] * subvects[threadIdx.y][threadIdx.x];
			__syncthreads();
			if (threadIdx.x < 16)  outmat[threadIdx.y][threadIdx.x] += outmat[threadIdx.y][threadIdx.x + 16];
			__syncthreads();
			if (threadIdx.x < 8)  outmat[threadIdx.y][threadIdx.x] += outmat[threadIdx.y][threadIdx.x + 8];
			__syncthreads();
			if (threadIdx.x < 4)  outmat[threadIdx.y][threadIdx.x] += outmat[threadIdx.y][threadIdx.x + 4];
			__syncthreads();
			if (threadIdx.x < 2)  outmat[threadIdx.y][threadIdx.x] += outmat[threadIdx.y][threadIdx.x + 2];
			__syncthreads();
			if (threadIdx.x == 0) {
				outmat[threadIdx.y][0] += outmat[threadIdx.y][1];
				out[shift] = outmat[threadIdx.y][0];
				shift += row + threadIdx.y - i - j;
			}				
			__syncthreads();
		}
	}
}

int main(int argc, char* argv[])
{
	//std::fstream input;
	//input.open("input", std::fstream::out);
	int Rows =  512;
	int Cols =  512;
	int BlocksX = Cols / BlckSzX;
	int BlocksY = Rows / BlckSzY;
	int slotSize = (Rows * (Rows + 1)) / 2;
	bool flag = true;

	float* vectors_h = genVects(Cols, Rows);
	float* out_vectors_h = new float[BlocksX * slotSize];
	float* means_h = new float[Rows];

		// Pointers for GPU (device) data
	float* vectors_d;
	float* means_d;
	float* out_vectors_d;

	// Safely allocate memory for data on device
	hipMalloc((void**)&vectors_d, Rows * Cols * sizeof(float));
	hipMalloc((void**)&means_d, Rows * BlocksX * sizeof(float));
	hipMalloc((void**)&out_vectors_d, sizeof(float) * slotSize * BlocksX );

	hipMemcpy(vectors_d, vectors_h, Cols * Rows * sizeof(float), hipMemcpyHostToDevice);

	// Split problem into threads
	dim3 blockGrid1(Cols/BlckSzX, Rows/BlckSzY,1);
	dim3 threadBlock1(BlckSzX,BlckSzY,1);
	mean_kern<<<blockGrid1, threadBlock1>>>(vectors_d, Rows, Cols, means_d, BlocksX);
	hipDeviceSynchronize();

	dim3 blockGrid2(Cols / BlckSzX, Rows / BlckSzY, 1);
	dim3 threadBlock2(BlckSzX, BlckSzY, 1);
	var_kern<<< blockGrid2, threadBlock2>>>(vectors_d, Rows, Cols, means_d, BlocksX);
	hipDeviceSynchronize();

	//hipMemcpy(means_h, means_d, Rows * sizeof(float), hipMemcpyDeviceToHost);
	//float* master_mean = findMeans(vectors_h, Cols, Rows);
	//for (int i = 0; i < Rows; ++i) flag = flag && (master_mean[i] == means_h[i]);

	dim3 blockGrid3(Cols / BlckSzX, Rows / BlckSzY, 1);
	dim3 threadBlock3(BlckSzX, BlckSzY, 1);
	VarianceMatrix <<<blockGrid3, threadBlock3 >>> (vectors_d, Rows, Cols, out_vectors_d, slotSize);
	
	hipDeviceSynchronize();
	hipMemcpy(out_vectors_h, out_vectors_d, BlocksX * slotSize * sizeof(float), hipMemcpyDeviceToHost);
	std::string str(hipGetErrorString(hipGetLastError()));

	std::vector<float> vec;
	float sum1 = 0.0f;
	for (int i = 0; i < BlocksX * slotSize; ++i) {
		sum1 += out_vectors_h[i];
		vec.push_back(out_vectors_h[i]);
	}

	sum1 /= (float)Rows;

	for (int i = 1; i < BlocksX; ++i) {
		for (int j = 0; j < slotSize; ++j) {
			out_vectors_h[j] += out_vectors_h[j + i * slotSize];
		}
	}
	for (int i = 0; i < slotSize; ++i) {
		out_vectors_h[i] /= (float)Rows;
	}

	float* out_vectors_h_master =  findCovVals(vectors_h, Cols, Rows);

	float minn = 0.0f;
	int cnnnt = 0;
	for (int i = 0; i < slotSize; ++i) {
		float ttt = std::abs(out_vectors_h[i] - out_vectors_h_master[i]);
		if (ttt > std::abs(minn)) minn = out_vectors_h[i] - out_vectors_h_master[i]; 
		if (ttt > 0.01f) cnnnt++;
	}//*/

	std::cout << "Min diff = " << minn << std::endl;

	float ssum = 0.0;
	for (int i = 0; i < slotSize; ++i) {
		ssum +=  out_vectors_h[i];
	}//*/

	hipFree(vectors_d);
	hipFree(means_d);
	hipFree(out_vectors_d);
}

float* genVects(int col, int row) {
	float* out = new float[col * row];
	for (int i = 0; i < col * row; ++i) {
		out[i] = 1.0;// static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
	}
	return out;
}

float* findMeans(float* vects, int col, int row) {
	float* means = new float[row];
	for (int i = 0; i < row; ++i) {
		float sum = 0.0f;
		float* subvects = &vects[col * i];
		for (int j = 0; j < col; ++j) {
			sum += subvects[j];
		}
		means[i] = 0;// sum / (float)col;
	}
	return means;
}

float* findCovVals(float* vects, int col, int row) {

	float* means = findMeans(vects, col, row);
	int sltsz = (row * (row + 1)) / 2;
	float* varmat = new float[sltsz];
	
	for (int i = 0; i < row; ++i) {
		int k = (i * (row + row + 1 - i)) / 2;
		for (int j = 0; (i + j) < row; ++j) {
			float sum = 0;
			for (int q = 0; q < col; ++q) {
				sum += (vects[j * col + q] - means[j]) * (vects[(j + i) * col + q] - means[j + i]);
			}
			varmat[k + j] = sum / (float)col;
		}
	}
	return varmat;
}